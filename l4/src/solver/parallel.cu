#include "hip/hip_runtime.h"
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#include "parallel.cuh"

#define errCheck(code) { errorCheck((code), __FILE__, __LINE__); }
void addWithCuda(const int * a, const int * b, int * c, int elements);

using std::cout;
using std::flush;
using std::endl;

inline void errorCheck(hipError_t code, const char* file, int line) {
    if(hipSuccess != code) {
        std::cout << "[" << file << ", line " << line << "]" << std::flush;
        std::cout << " CUDA error <" << hipGetErrorString(code) << "> received." << std::endl << std::flush;
        exit(EXIT_FAILURE);
    }
}

__global__ void addKernel(const int * a, const int * b, int * c, int elements) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    printf("x = %d, BlockIdx = %d, BlockDimx = %d, threadIdx = %d\n", x, blockIdx.x, blockDim.x, threadIdx.x);
    if(x < elements) {
        c[x] = a[x] + b[x];
    }
}

void solvePar(int rows, int cols, int iterations, double td, double h, double ** matrix) {
    cout << "Do cuda related stuff here!" << endl << flush;

    // Example.
    int elements = 5;
    const int a[elements] = { 1, 2, 3, 4, 5 }; 
    const int b[elements] = { 5, 4, 3, 2, 1 }; 
    int c[elements] = { 0 }; 

    addWithCuda(a, b, c, elements);
}

void addWithCuda(const int * a, const int * b, int * c, int elements) {
    int * dev_a = nullptr;
    int * dev_b = nullptr;
    int * dev_c = nullptr;

    dim3 dimGrid(1, 1, 1);
    dim3 dimBlock(5, 1, 1);

    errCheck(hipSetDevice(0));

    errCheck(hipMalloc((void**) &dev_c, elements * sizeof(int)));
    errCheck(hipMalloc((void**) &dev_a, elements * sizeof(int)));
    errCheck(hipMalloc((void**) &dev_b, elements * sizeof(int)));

    errCheck(hipMemcpy(dev_a, a, elements * sizeof(int), hipMemcpyHostToDevice));
    errCheck(hipMemcpy(dev_b, b, elements * sizeof(int), hipMemcpyHostToDevice));

    addKernel<<<dimGrid, dimBlock>>>(dev_a, dev_b, dev_c, elements);

    errCheck(hipGetLastError());
    errCheck(hipDeviceSynchronize());
    errCheck(hipMemcpy(c, dev_c, elements * sizeof(int), hipMemcpyDeviceToHost));
    errCheck(hipFree(dev_a));
    errCheck(hipFree(dev_b));
    errCheck(hipFree(dev_c));
    errCheck(hipDeviceReset());

    cout << "c = { " << c[0] << flush;
    for(int i = 1; i < elements; i++) {
        cout << ", " << c[i] << flush;
    }

    cout << " }" << endl << flush;
}
